#include "hip/hip_runtime.h"
#include <nccl.h>
#include <cstdio>
#include <cstdlib>
 
__global__ void kernel(int npointslocal) 
{
    addlocal = 0;
    
  int index = threadIdx.x;

  for (i = 1; i <= npointslocal; ++i){
    addlocal = addlocal + 1.0/(1+x*x);
    x = x + delta;
  }

}/*kernel*/
 

void print_vector(int *in, int n){

 for(int i=0; i < n; i++)
  printf("%d\t", in[i]);

 printf("\n");

}/*print_vector*/


int main(int argc, char* argv[]) {

  int data_size = 1 ;
  int nGPUs = 0;
  hipGetDeviceCount(&nGPUs);
  
  int master = 0, size, myrank, npoints, npointslocal, i;
  double delta, add, addlocal, x;

  printf("Numbers of divide points:");
  scanf("%ld", &npoints);

  int *DeviceList = (int *) malloc (nGPUs     * sizeof(int));
  int *data       = (int*)  malloc (data_size * sizeof(int));
  int **d_data    = (int**) malloc (nGPUs     * sizeof(int*));

  data = npoints;
  
  for(int i = 0; i < nGPUs; i++)
      DeviceList[i] = i;
  
  /*Initializing NCCL with Multiples Devices per Thread*/
  ncclComm_t* comms = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * nGPUs);  
  hipStream_t* s   = (hipStream_t*)malloc(sizeof(hipStream_t)* nGPUs);
  ncclCommInitAll(comms, nGPUs, DeviceList);
      
  for(int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      hipStreamCreate(&s[g]);
      hipMalloc(&d_data[g], data_size * sizeof(int));
     
      if(g == 0)  /*Copy from Host to Device*/
         hipMemcpy(d_data[g], data, data_size * sizeof(int), hipMemcpyHostToDevice);
  }
        
  ncclGroupStart();
 
  		for(int g = 0; g < nGPUs; g++) {
  	  	    hipSetDevice(DeviceList[g]);
    	  	    ncclBcast(d_data[g], data_size, ncclInt, 0, comms[g], s[g]); /*Broadcasting it to all*/
  		}

  ncclGroupEnd();       

  for (int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      printf("\nThis is device %d\n", g);
      kernel <<< 1 , data_size >>> (d_data[g]);/*Call the CUDA Kernel: The code multiple the vector position per 2 on GPUs*/
      hipDeviceSynchronize();               
  }

  printf("\n");

  for (int g = 0; g < nGPUs; g++) { /*Synchronizing CUDA Streams*/
      hipSetDevice(DeviceList[g]);
      hipStreamSynchronize(s[g]);
  }
 
  for(int g = 0; g < nGPUs; g++) {  /*Destroy CUDA Streams*/
      hipSetDevice(DeviceList[g]);
      hipStreamDestroy(s[g]);
  }

  for(int g = 0; g < nGPUs; g++)    /*Finalizing NCCL*/
     ncclCommDestroy(comms[g]);
  
  /*Freeing memory*/
  free(s);
  free(data); 
  free(DeviceList);

  hipFree(d_data);

  return 0;

}/*main*/