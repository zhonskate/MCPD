
/*************************************
 * Matrix-Vector product CUDA kernel *
 * V2: With Shared memory            *
 *************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_SAFE_CALL( call ) {                                         \
 hipError_t err = call;                                                 \
 if( hipSuccess != err ) {                                              \
   fprintf(stderr,"CUDA: error occurred in cuda routine. Exiting...\n"); \
   exit(err);                                                            \
 } }

#define	A(i,j)		A[ (j) + ((i)*(n)) ]
#define	x(i) 		x[ (i) ]
#define	y(i) 		y[ (i) ]
#define	y_gpu(i) 	y_gpu[ (i) ]
#define	y_cpu(i) 	y_cpu[ (i) ]
#define	d_A(i,j) 	d_A[ (j) + ((i)*(n)) ]
#define	d_x(i) 		d_x[ (i) ]
#define	d_y(i) 		d_y[ (i) ]

#define min(i,j)	( (i)<(j) ? (i) : (j) )

#define	BLOCKSIZE	32

__global__ void compute_kernel( unsigned int m, unsigned int n, float *d_A, float *d_x, float *d_y ) {
  
  /* Obtain (x,y) coordinates of the thread within the block */

  /* Obtain the global index to a matrix row (variable i). Note that there is only one dimension in the grid
     so blockIdx.x and blockDim.x are the only existing variables */
  unsigned int i;
  unsigned int j;
  i = blockIdx.x * BLOCKSIZE;
  j = blockIdx.y * BLOCKSIZE;

  __syncthreads();


  /* Declare share memory space of a piece of array d_x of size BLOCKSIZE */
  __shared__ float b[BLOCKSIZE];
  /* Declare share memory space of a square block of order BLOCKSIZE */
  __shared__ float a[BLOCKSIZE*BLOCKSIZE];
  
  __syncthreads();


  if( i < m ) { /* Prevent work on positions beyond m */
 
    b[threadIdx.x] = d_x[i + threadIdx.x];
    __syncthreads();

     /* Implement Part 1 here */
     /* Loop (threadIdx.x:BLOCKSIZE:n-1) */
        /* Copy subvector x in shared memory */
        /* Perform the add+product on a local variable */
      float sum = 0.0f;
      for(int k=threadIdx.x;k<n-1;k+=BLOCKSIZE){
        sum += d_A(j,k)*b[k];
      }

     /* Save local variable in shared memory */
     a[i+BLOCKSIZE*j] = sum;

     /* Implement Part 2 here */
     /* Only if threadIdx.x==0 */
     if(threadIdx.x==0){
       for(int l = 0;l<BLOCKSIZE;l++){
         d_y(i)+=a[i + BLOCKSIZE*l];
       }
     }

       /* Add all column elements along a row of the shared memory square block */
       /* Save result in d_y */

  }
}

int cu_matrix_vector( unsigned int m, unsigned int n, float *h_A, float *h_x, float *h_y ) {

  // Allocate device memory
  float *d_A, *d_x, *d_y;
  CUDA_SAFE_CALL( hipMalloc((void **) &d_A, m*n*sizeof(float) ) );
  CUDA_SAFE_CALL( hipMalloc((void **) &d_x,   n*sizeof(float) ) );
  CUDA_SAFE_CALL( hipMalloc((void **) &d_y, m  *sizeof(float) ) );

  // Copy host memory to device 
  CUDA_SAFE_CALL( hipMemcpy( d_A, h_A, m*n*sizeof(float), hipMemcpyHostToDevice ) );
  CUDA_SAFE_CALL( hipMemcpy( d_x, h_x,   n*sizeof(float), hipMemcpyHostToDevice ) );

  int n_blocks = (int) ceil( (float) m / (float) BLOCKSIZE );

  // Execute the kernel
  dim3 dimGrid( n_blocks );
  dim3 dimBlock( BLOCKSIZE, BLOCKSIZE );
  compute_kernel<<< dimGrid, dimBlock >>>( m, n, d_A, d_x, d_y );

  // Copy device memory to host 
  CUDA_SAFE_CALL( hipMemcpy( h_y, d_y, m  *sizeof(float), hipMemcpyDeviceToHost ) );

  // Deallocate device memory
  CUDA_SAFE_CALL( hipFree(d_A) );
  CUDA_SAFE_CALL( hipFree(d_x) );
  CUDA_SAFE_CALL( hipFree(d_y) );

  return EXIT_SUCCESS;
}
 
int matrix_vector( unsigned int m, unsigned int n, float *A, float *x, float *y ) {

  unsigned int i, j;
  for( i=0; i<m; i++ ) {
    y( i ) = 0.0f;
    for( j=0; j<n; j++ ) {
      y( i ) += A( i, j ) * x( j );
    }
  }
  return EXIT_SUCCESS;

}

int main( int argc, char *argv[] ) {
  unsigned int m, n;
  unsigned int i, j;

  /* Generating input data */
  if( argc<3 ) {
    printf("Usage: %s n_rows n_cols \n",argv[0]);
    exit(-1);
  }
  sscanf(argv[1],"%d",&m);
  sscanf(argv[2],"%d",&n);
  float *A = (float *) malloc( m*n*sizeof(float) );
  float *x = (float *) malloc(   n*sizeof(float) );
  printf("%s: Generating a random matrix of size %dx%d and a vector of size %d...\n",argv[0],m,n,n);
  for( i=0; i<m; i++ ) {
    for( j=0; j<n; j++ ) {
      A( i, j ) = 2.0f * ( (float) rand() / RAND_MAX ) - 1.0f;
    }
  }
  for( j=0; j<n; j++ ) {
    x( j ) = 2.0f * ( (float) rand() / RAND_MAX ) - 1.0f;
  }


  // Allocate CUDA events that we'll use for timing
  hipEvent_t start, stop;
  CUDA_SAFE_CALL( hipEventCreate(&start) );
  CUDA_SAFE_CALL( hipEventCreate(&stop) );

  printf("%s: y=A*x in CPU...\n",argv[0]);
  float *y_cpu = (float *) malloc( m*sizeof(float) );
  CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event
  matrix_vector( m, n, A, x, y_cpu );
  CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  float msecCPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecCPU, start, stop) );

  printf("%s: y=A*x in GPU...\n",argv[0]);
  float *y_gpu = (float *) malloc( m*sizeof(float) );
  CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event
  cu_matrix_vector( m, n, A, x, y_gpu );
  CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  float msecGPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecGPU, start, stop) );

  /* Check for correctness */
  float max = fabs( y_cpu( 0 ) );
  for( i=1; i<m; i++ ) {
    max = fabs( y_cpu( i ) > max ? y_cpu( i ) : max );
  }
  float error = 0.0f;
  for( i=0; i<m; i++ ) {
    error += fabs( y_gpu( i ) - y_cpu( i ) );
  }
  printf("Error CPU/GPU = %.3e\n",error/max);
  double flops = 2.0 * (double) m * (double) n;
  double gigaFlopsCPU = (flops * 1.0e-9f) / (msecCPU / 1000.0f);
  double gigaFlopsGPU = (flops * 1.0e-9f) / (msecGPU / 1000.0f);
  printf("CPU time = %.2f msec.\n",msecCPU);
  printf("GPU time = %.2f msec.\n",msecGPU);
  printf("Gflops CPU = %.2f \n",gigaFlopsCPU);
  printf("Gflops GPU = %.2f \n",gigaFlopsGPU);
  
  free(A);
  free(x);
  free(y_cpu);
  free(y_gpu);
  
}

