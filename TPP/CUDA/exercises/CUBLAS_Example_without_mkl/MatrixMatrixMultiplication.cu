
/*************************************
 * Matrix-Matrix product with CUBLAS *
 *************************************/

#include <stdio.h>
#include <...> /* Write here the name of the CUBLAS header file */

#define CUDA_SAFE_CALL( call ) {                                         \
 hipError_t err = call;                                                 \
 if( hipSuccess != err ) {                                              \
   fprintf(stderr,"CUDA: error occurred in cuda routine. Exiting...\n"); \
   exit(err);                                                            \
 } }
#define CUBLAS_SAFE_CALL( call ) {                                         \
 hipblasStatus_t err = call;                                                \
 if( HIPBLAS_STATUS_SUCCESS != err ) {                                      \
   fprintf(stderr,"CUBLAS: error occurred in cuda routine. Exiting...\n"); \
   hipblasDestroy(handle);                                                  \
   exit(err);                                                              \
 } }

/* Matrices stored by columns: BLAS style */
#define	A(i,j)		A[ (i) + ((j)*(n)) ]
#define	B(i,j)		B[ (i) + ((j)*(n)) ]
#define	C(i,j)		C[ (i) + ((j)*(n)) ]
#define	gpu_C(i,j)	gpu_C[ (i) + ((j)*(n)) ]
#define	d_A(i,j) 	d_A[ (j) + ((i)*(n)) ]

int main( int argc, char *argv[] ) {
  int n;
  unsigned int i, j;

  /* Generating input data */
  if( argc<2 ) {
    printf("Usage: %s n \n",argv[0]);
    exit(-1);
  }
  sscanf(argv[1],"%d",&n);
  double *A = (double *) malloc( n*n*sizeof(double) );
  double *B = (double *) malloc( n*n*sizeof(double) );
  double *C = (double *) malloc( n*n*sizeof(double) );
  double *gpu_C = (double *) malloc( n*n*sizeof(double) );
  printf("%s: Generating two random matrices of size %dx%d...\n",argv[0],n,n);
  for( i=0; i<n; i++ ) {
    for( j=0; j<n; j++ ) {
      A( i, j ) = 2.0 * ( (double) rand() / RAND_MAX ) - 1.0;
    }
  }
  for( i=0; i<n; i++ ) {
    for( j=0; j<n; j++ ) {
      B( i, j ) = 2.0 * ( (double) rand() / RAND_MAX ) - 1.0;
    }
  }

  /* STARTUP CUBLAS context */
  /* Declare a CUBLAS handle with name handle */
  . . . 
  CUBLAS_SAFE_CALL( /* Create cublas handle */ );

  // Allocate CUDA events that we'll use for timing
  hipEvent_t start, stop;
  CUDA_SAFE_CALL( hipEventCreate(&start) );
  CUDA_SAFE_CALL( hipEventCreate(&stop) );

  const double ONE = 1.0;
  const double ZERO = 0.0;

  printf("%s: C=A*B in GPU...\n",argv[0]);
  // Allocate device memory
  double *d_A, *d_B, *d_C;
  CUDA_SAFE_CALL( hipMalloc((void **) &d_A, n*n*sizeof(double) ) );
  CUDA_SAFE_CALL( hipMalloc((void **) &d_B, n*n*sizeof(double) ) );
  CUDA_SAFE_CALL( hipMalloc((void **) &d_C, n*n*sizeof(double) ) );
  CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event
  /* In this place transfer matrices A and B from Host to matrices d_A and d_B, respectively, on Device */
  CUDA_SAFE_CALL( /* A -> d_A */ );
  CUDA_SAFE_CALL( /* B -> d_B */ );
  CUBLAS_SAFE_CALL( /* In this place write a call to routine hipblasDgemm that performs the matrix-matrix product on GPU */ );
  /* In this place transfer matrix d_C from Device to matrix gpu_C on the Host */
  CUDA_SAFE_CALL( /* d_C -> gpu_C */ );
  CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  float msecGPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecGPU, start, stop) );

  double flops = 2.0 * (double) n * (double) n * (double) n;
  float gigaFlopsGPU = (flops * 1.0e-9f) / (msecGPU / 1000.0f);
  printf("GPU time = %.2f msec.\n",msecGPU);
  printf("Gflops GPU = %.2f \n",gigaFlopsGPU);
  
  /* Destroy the CUBLAS handle by calling routine hipblasDestroy */
  free(A);
  free(B);
  free(C);
  free(gpu_C);
  
}

