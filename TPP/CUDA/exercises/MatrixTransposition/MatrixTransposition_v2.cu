
/****************************************
 * CUDA kernel for transposing matrices *
 * Version with Shared Memory           *
 ****************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_SAFE_CALL( call ) {                                         \
 hipError_t err = call;                                                 \
 if( hipSuccess != err ) {                                              \
   fprintf(stderr,"CUDA: error occurred in cuda routine. Exiting...\n"); \
   exit(err);                                                            \
 } }

#define	A(i,j)		A[ (j) + ((i)*(n)) ]
#define	B(i,j)		B[ (j) + ((i)*(m)) ]
#define	B_cpu(i,j) 	B_cpu[ (j) + ((i)*(m)) ]
#define	B_gpu(i,j) 	B_gpu[ (j) + ((i)*(m)) ]
#define	d_A(i,j) 	d_A[ (j) + ((i)*(n)) ]
#define	d_B(i,j) 	d_B[ (j) + ((i)*(m)) ]
#define	sh_A(i,j) 	sh_A[ (i) ][ (j) ]

#define	BLOCKSIZE	16

/* Inefficient kernel: bad access to memory */
__global__ void compute_kernel( unsigned int m, unsigned int n, float *d_A, float *d_B ) {
    /* Index of thread in x dimension */
    /* Index of thread in y dimension */
    /* Index of block in x dimension */ 
    /* Index of block in y dimension */ 
    /* Global index to a row of A */
    /* Global index to a col of A */
    /* Global index to a row of B */
    /* Global index to a col of B */

    /* Declare a shared memory tile of size BLOCKSIZExBLOCKSIZE */

    /* Copy Element d_A( r_A, c_A ) to shared memory and 
       from shared_memory to element d_B( r_B, c_B ) 
       Prevent access to non-existing elements of A
       Remember synchronize threads upon data has been saved into shared memory */

}

int cu_transpose( unsigned int m, unsigned int n, float *h_A, float *h_B  ) {

  // Allocate device memory
  unsigned int mem_size = m * n * sizeof(float);
  float *d_A, *d_B;
  CUDA_SAFE_CALL( hipMalloc((void **) &d_A, mem_size ) );
  CUDA_SAFE_CALL( hipMalloc((void **) &d_B, mem_size ) );

  // Copy host memory to device 
  CUDA_SAFE_CALL( hipMemcpy( d_A, h_A, mem_size, hipMemcpyHostToDevice ) );

  int blocks_col = (int) ceil( (float) n / (float) BLOCKSIZE );
  int blocks_row = (int) ceil( (float) m / (float) BLOCKSIZE );

  // Execute the kernel
  dim3 dimGrid( blocks_col, blocks_row );
  dim3 dimBlock( BLOCKSIZE, BLOCKSIZE );
  compute_kernel<<< dimGrid, dimBlock >>>( m, n, d_A, d_B );

  // Copy device memory to host 
  CUDA_SAFE_CALL( hipMemcpy( h_B, d_B, mem_size, hipMemcpyDeviceToHost ) );

  // Deallocate device memory
  CUDA_SAFE_CALL( hipFree(d_A) );
  CUDA_SAFE_CALL( hipFree(d_B) );

  return EXIT_SUCCESS;
}
 
int transpose( unsigned int m, unsigned int n, float *A, float *B ) {

  unsigned int i, j;
  for( i=0; i<m; i++ ) {
    for( j=0; j<n; j++ ) {
      B( j, i ) = A( i, j );
    }
  }
  return EXIT_SUCCESS;

}

void printMatrix( unsigned int m, unsigned int n, float *A ) {
  int i, j;
  for( i=0; i<m; i++ ) {
    for( j=0; j<n; j++ ) {
      printf("%8.1f",A(i,j));
    }
    printf("\n");
  }
}

int main( int argc, char *argv[] ) {
  unsigned int m, n;
  unsigned int i, j;

  /* Generating input data */
  if( argc<3 ) {
    printf("Usage: %s n_rows n_cols \n",argv[0]);
    exit(-1);
  }
  sscanf(argv[1],"%d",&m);
  sscanf(argv[2],"%d",&n);
  float *A = (float *) malloc( m*n*sizeof(float) );
  printf("%s: Generating a random matrix of size %dx%d...\n",argv[0],m,n);
  for( i=0; i<m; i++ ) {
    for( j=0; j<n; j++ ) {
      A( i, j ) = 2.0f * ( (float) rand() / RAND_MAX ) - 1.0f;
    }
  }
  float *B_cpu = (float *) malloc( m*n*sizeof(float) );
  float *B_gpu = (float *) malloc( m*n*sizeof(float) );

  printf("%s: Transposing matrix A into B in CPU...\n",argv[0]);
  transpose( m, n, A, B_cpu );

  printf("%s: Transposing matrix A into B in GPU...\n",argv[0]);
  cu_transpose( m, n, A, B_gpu );

  /* Check for correctness */
  float error = 0.0f;
  for( i=0; i<n; i++ ) {
    for( j=0; j<m; j++ ) {
      error += fabs( B_gpu( i, j ) - B_cpu( i, j ) );
    }
  }
  printf("Error CPU/GPU = %.3e\n",error);
  
  free(A);
  free(B_cpu);
  free(B_gpu);
  
}

